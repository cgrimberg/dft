#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <ctype.h>


#define TOTALELEMENTS 2048//(64*1024)
#define SAMPLINGFREQ   1024
#define PI 			   3.14159

hipDoubleComplex iii = make_hipDoubleComplex(0, 1);

hipDoubleComplex * ListO;//[TOTALELEMENTS];
hipDoubleComplex * GPUListSrc;//[TOTALELEMENTS];
hipDoubleComplex * GPUListDst;//[TOTALELEMENTS];
hipDoubleComplex * ListD;//[TOTALELEMENTS];

struct pair
{
	hipDoubleComplex mag;
	double freq;
	double phase;
};

struct pair * signalList;
struct pair * GPUsignalList;

__host__ void printContents(hipDoubleComplex * a, int start, int count)
{
	for (int i = 0; i < count; i++)
	{
		printf("%f + %fi\t", a[start+i].x, a[start+i].y);
	}
	printf("\n");
	return;
}

__device__ __host__ hipDoubleComplex cuCpow(hipDoubleComplex a, int b)
{
	hipDoubleComplex o = make_hipDoubleComplex(1, 0);
	for (int i = 0; i < b; i++)
	{
		o = hipCmul(o, a);
	}
	return o;
}

__global__
void fft(hipDoubleComplex *src, hipDoubleComplex *dst, int length, hipDoubleComplex rou)
{
	unsigned int ThrPerBlk = blockDim.x;
	unsigned int MYbid = blockIdx.x;
	unsigned int MYtid = threadIdx.x;
	unsigned int MYgtid = ThrPerBlk * MYbid + MYtid;
	hipDoubleComplex wn1 = cuCpow(rou, MYgtid);
	hipDoubleComplex wnn = make_hipDoubleComplex(1,0);
	int i;
	for (i = 0; i < length; i++)
	{
		dst[MYgtid] = hipCadd(dst[MYgtid], hipCmul(src[i],wnn));
		wnn = hipCmul(wnn, wn1);
	}
}

__global__
void ifft(hipDoubleComplex *src, hipDoubleComplex *dst, int length, hipDoubleComplex rou)
{
	unsigned int ThrPerBlk = blockDim.x;
	unsigned int MYbid = blockIdx.x;
	unsigned int MYtid = threadIdx.x;
	unsigned int MYgtid = ThrPerBlk * MYbid + MYtid;
	hipDoubleComplex wn1 = cuCpow(rou, MYgtid);
	hipDoubleComplex wnn = make_hipDoubleComplex(1,0);

	int i;
	for (i = 0; i < length; i++)
	{
		dst[MYgtid] = hipCadd(dst[MYgtid], hipCdiv(src[i], wnn));
		wnn = hipCmul(wnn, wn1);
	}
	dst[MYgtid] = hipCdiv(dst[MYgtid],make_hipDoubleComplex(length,0));
}

__global__
void create(pair * signals, int length, int random, hipDoubleComplex * dst)
{
    int i;
	pair item;
	unsigned int ThrPerBlk = blockDim.x;
	unsigned int MYbid = blockIdx.x;
	unsigned int MYtid = threadIdx.x;
	unsigned int MYgtid = ThrPerBlk * MYbid + MYtid;
	float t = ((float)MYgtid)/SAMPLINGFREQ;
    for(i=0; i<length; i++)
    {
        item = signals[i];
		dst[MYgtid] = hipCadd(dst[MYgtid], hipCmul(item.mag, make_hipDoubleComplex((double)(cosf(item.freq*2*PI*t + item.phase)), (double)(sinf(item.freq*2*PI*t + item.phase)))));
    }
	//dst[MYgtid] = cuCadd(dst[MYgtid], make_cuDoubleComplex((rand() % random ) * ((rand() % 2)*2-1),0));
}

double mse(hipDoubleComplex * numb1, hipDoubleComplex * numb2, int elements)
{
    int i;
	double error = 0;
    for(i=0;i<elements;i++)
    {
        error += hipCabs(cuCpow(hipCsub(numb1[i],numb2[i]), 2));
    }
    return error/elements;

}

int main(int argc, char **argv)
{
	srand(time(NULL));


	ListO = (hipDoubleComplex *) malloc(TOTALELEMENTS*sizeof(hipDoubleComplex));
	if (ListO==NULL)
	{
		printf("Cannot allocate memory for NumebrsO.\n");
		exit(EXIT_FAILURE);
	}
    ListD = (hipDoubleComplex *) malloc(TOTALELEMENTS*sizeof(hipDoubleComplex));
	if (ListD==NULL)
	{
		free(ListO);
		printf("Cannot allocate memory for NumebrsQ.\n");
		exit(EXIT_FAILURE);
	}


	// max of 10 sinusoids
	signalList = (pair *) malloc(10*sizeof(pair));
	if (signalList==NULL)
	{
		free(ListD);
		free(ListO);
		printf("Cannot allocate memory for NumebrsQ.\n");
		exit(EXIT_FAILURE);
	}

	////////////////////////////// define sinusoids hre.
	int sinusoidCount = 5;
	signalList[0].mag = make_hipDoubleComplex(8, 0);
	signalList[0].freq = 0;
	signalList[0].phase = 0;
	
	signalList[1].mag = make_hipDoubleComplex(0,-3.5);
	signalList[1].freq = 50;
	signalList[1].phase = 0;

	signalList[2].mag = make_hipDoubleComplex(0,3.5);
	signalList[2].freq = -50;
	signalList[2].phase = 0;

	signalList[3].mag = make_hipDoubleComplex(0,-5);
	signalList[3].freq = 120;
	signalList[3].phase = 0;

	signalList[4].mag = make_hipDoubleComplex(0,5);
	signalList[4].freq = -120;
	signalList[4].phase = 0;
	int noiseMagnitude = 20;
	//printf("%f\n\n\n", signalList[1].mag*cosf(signalList[1].freq*2*PI*0.250 + signalList[1].phase));

	//float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t cudaStatus, cudaStatus2, cudaStatus3;
	//char InputFileName[255], OutputFileName[255], ProgName[255];
	int ThrPerBlk=256;
	int NumBlocks=TOTALELEMENTS/ThrPerBlk;
	hipDeviceProp_t GPUprop;
	//unsigned long SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;		char SupportedBlocks[100];

	// Choose which GPU to run on, change this on a multi-GPU system.
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("\nNo CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(EXIT_FAILURE);
	}
	hipGetDeviceProperties(&GPUprop, 0);
	// SupportedKBlocks = (unsigned int)GPUprop.maxGridSize[0] * (unsigned int)GPUprop.maxGridSize[1] * (unsigned int)GPUprop.maxGridSize[2] / 1024;
	// SupportedMBlocks = SupportedKBlocks / 1024;
	// sprintf(SupportedBlocks, "%lu %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
	// MaxThrPerBlk = (unsigned int)GPUprop.maxThreadsPerBlock;


	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((void**)&GPUListSrc, TOTALELEMENTS*sizeof(hipDoubleComplex));
	cudaStatus2 = hipMalloc((void**)&GPUListDst, TOTALELEMENTS*sizeof(hipDoubleComplex));
	cudaStatus3 = hipMalloc((void**)&GPUsignalList, TOTALELEMENTS*sizeof(pair));
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess) || (cudaStatus3 != hipSuccess)){
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory");
		exit(EXIT_FAILURE);
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPUsignalList, signalList, TOTALELEMENTS*sizeof(pair), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!");
		exit(EXIT_FAILURE);
	}


	create <<< NumBlocks, ThrPerBlk >>> (GPUsignalList, sinusoidCount, noiseMagnitude, GPUListSrc);

	printf("Finish create\n");
	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\nhipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipMemcpy(ListO, GPUListSrc, TOTALELEMENTS*sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!");
		exit(EXIT_FAILURE);
	}
	//printContents(ListO, 0, 20);

	hipDoubleComplex rou = make_hipDoubleComplex(cosf(-2*PI/TOTALELEMENTS), sinf(-2*PI/TOTALELEMENTS));
	fft <<< NumBlocks, ThrPerBlk >>> (GPUListSrc, GPUListDst, TOTALELEMENTS, rou);
	
	cudaStatus = hipDeviceSynchronize();
	//checkError(cudaGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
		free(ListD);
		free(ListO);
		free(signalList);
		exit(EXIT_FAILURE);
	}

	printf("Finish fft\n");

/**************** delete later 
	cudaStatus = cudaMemcpy(ListO, GPUListDst, TOTALELEMENTS*sizeof(float), cudaMemcpyDeviceToHost);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaMemcpy GPU to CPU  failed!");
		exit(EXIT_FAILURE);
	}
	printContents(ListO, 235, 10);
*****/

	ifft <<< NumBlocks, ThrPerBlk >>> (GPUListDst, GPUListSrc, TOTALELEMENTS, rou);

	cudaStatus = hipDeviceSynchronize();
	//checkError(cudaGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
		free(ListD);
		free(ListO);
		free(signalList);
		exit(EXIT_FAILURE);
	}
	printf("Finish ifft\n");
	cudaStatus = hipMemcpy(ListD, GPUListSrc, TOTALELEMENTS*sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!");
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipDeviceSynchronize();
	//checkError(cudaGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
		free(ListD);
		free(ListO);
		free(signalList);
		exit(EXIT_FAILURE);
	}

	printf("Mean Squared Error: %f\n", mse(ListO, ListD, TOTALELEMENTS));

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(GPUListDst);
	hipFree(GPUListSrc);
	hipFree(GPUsignalList);
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(ListD);
		free(ListO);
		free(signalList);
		exit(EXIT_FAILURE);
	}
	free(ListD);
	free(ListO);
	free(signalList);
	return(EXIT_SUCCESS);
}



